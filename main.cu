#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <GL/gl.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <cuda_surface_types.h>
#include <hip/hip_texture_types.h>
#include <>
#include <cuda_device_runtime_api.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>
#include <cutil.h>
#include <omp.h>
#include <helper_gl.h> 
#include <hip/hip_vector_types.h>

using namespace std;

#pragma pack(1)
#define Tx 16
#define Ty 16

#define MAX_EPSILON_ERROR   5.0f
#define REFRESH_DELAY       10 
#define MIN_EUCLIDEAN_D     0.01f
#define MAX_EUCLIDEAN_D     5.f
#define MAX_FILTER_RADIUS   25
#define GL_TEXTURE_TYPE GL_TEXTURE_2D


//cuda graphics resources
struct uchar4;
GLuint pbo;
GLuint texid;  
GLuint shader;
struct hipGraphicsResource *cuda_pbo_resource;


//Timer counter
StopWatchInterface *timer = NULL;
StopWatchInterface *kernel_timer = NULL;

//Parameters
float gaussian_delta = 4;
float euclidean_delta = 0.1f;
const char **pArgv = NULL;
unsigned int *pArgc = NULL;
unsigned int iterations = 1;
unsigned int filter_radius = 5;
unsigned int width;
unsigned int height;
unsigned int  *hImage  = NULL;
unsigned int fpsCount = 0;
unsigned int fpsLimit = 1;
unsigned int g_TotalErrors = 0;
unsigned int devID=0;
unsigned int  dev;
bool g_bInteractive = false;


//BMP Data structure
typedef struct
{
    short type;
    int size;
    short reserved1;
    short reserved2;
    int offset;
} BMPHeader;

typedef struct
{
    int size;
    int width;
    int height;
    short planes;
    short bitsPerPixel;
    unsigned compression;
    unsigned imageSize;
    int xPelsPerMeter;
    int yPelsPerMeter;
    int clrUsed;
    int clrImportant;
} BMPInfoHeader;



//Function prototype;
static double imageFilterRGBA(unsigned int *d_dest, int width, int height,float e_d, 
        int radius, int iterations, StopWatchInterface *timer);
static void updateGaussian(float delta, int radius);
static void LoadBMPFile(uchar4 **dst, unsigned int *width, unsigned int *height, const char *name);



__constant__ float cGaussian[64]; 
hipTextureObject_t rgbaTexdImage;
hipTextureObject_t rgbaTexdTemp;

uint *dImage  = NULL;  
uint *dTemp   = NULL;   
size_t pitch;


//Euclidean Distance measure
__device__ float euclideanLen(float4 a, float4 b, float d)
{

    float mod = (b.x - a.x) * (b.x - a.x) +  (b.y - a.y) * (b.y - a.y) +  (b.z - a.z) * (b.z - a.z);

    return (__expf(-mod / (2.f * d * d)));
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(fabs(rgba.x)); 
    rgba.y = __saturatef(fabs(rgba.y));
    rgba.z = __saturatef(fabs(rgba.z));
    rgba.w = __saturatef(fabs(rgba.w));
    return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) 
            | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}

__device__ float4 rgbaIntToFloat(uint c)
{
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f; 
    rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  
    rgba.z = ((c>>16) & 0xff) * 0.003921568627f; 
    rgba.w = ((c>>24) & 0xff) * 0.003921568627f; 
    return rgba;
}

//Kernel for filter 
__global__ void
rgbImagefilter(uint *od, int w, int h,  float e_d,  int r, hipTextureObject_t rgbaTex)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int i= y * w + x;

    if (x >= w || y >= h)
    {
        return;
    }

    float sum = 0.0f;
    float factor;
    float4 t = {0.f, 0.f, 0.f, 0.f};
    float4 center = tex2D<float4>(rgbaTex, x, y);

    for (int i = -r; i <= r; i++){
        for (int j = -r; j <= r; j++){
            float4 curPix = tex2D<float4>(rgbaTex, x + j, y + i);
            factor = cGaussian[i + r] * cGaussian[j + r] * euclideanLen(curPix, center, e_d); 
            t += factor * curPix;
            sum += factor;
        }
    }
    od[i] = rgbaFloatToInt(t/sum);
}


static void initTexture(int width, int height, uint *hImage){
    // copy image data to cuda array
    hipMallocPitch(&dImage, &pitch, sizeof(uint)*width, height);
    hipMallocPitch(&dTemp,  &pitch, sizeof(uint)*width, height);
    hipMemcpy2D(dImage, pitch, hImage, sizeof(uint)*width,sizeof(uint)*width, height,hipMemcpyHostToDevice);
    hipStreamSynchronize(0);
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    hipResourceDesc  texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType  = hipResourceTypePitch2D;
    texRes.res.pitch2D.devPtr   = dImage;
    texRes.res.pitch2D.desc     = desc;
    texRes.res.pitch2D.width    = width;
    texRes.res.pitch2D.height   = height;
    texRes.res.pitch2D.pitchInBytes = pitch;
    hipTextureDesc   texDescr;
    memset(&texDescr,0,sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode   = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeNormalizedFloat;

    hipCreateTextureObject(&rgbaTexdImage, &texRes, &texDescr, NULL);
    memset(&texRes,0,sizeof(hipResourceDesc));

    texRes.resType   = hipResourceTypePitch2D;
    texRes.res.pitch2D.devPtr   = dTemp;
    texRes.res.pitch2D.desc     = desc;
    texRes.res.pitch2D.width    = width;
    texRes.res.pitch2D.height   = height;
    texRes.res.pitch2D.pitchInBytes = pitch;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode   = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeNormalizedFloat;
    hipCreateTextureObject(&rgbaTexdTemp, &texRes, &texDescr, NULL);
    
}


static void updateGaussian(float delta, int radius){
    float  fGaussian[64];
    #pragma omp parallel  for
    for (int i = 0; i < 2*radius + 1; ++i)
    {
        float x = i-radius;
        fGaussian[i] = expf(-(x*x) / (2*delta*delta));
    }
    hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float)*(2*radius+1));
    hipStreamSynchronize(0);
}

static double imageFilterRGBA(uint *dDest, int width, int height, float e_d, int radius, 
        int iterations, StopWatchInterface *timer)
{
    double dKernelTime;
    for (int i=0; i<iterations; i++)
    {
        dKernelTime = 0.0;
        hipDeviceSynchronize();
        sdkResetTimer(&timer);
        dim3 gridSize((width + Tx - 1) / Ty, (height + Ty - 1) / Ty);
        dim3 blockSize(Tx, Ty);
        rgbImagefilter<<< gridSize, blockSize>>>(dDest, width, height, e_d, radius, rgbaTexdImage);       
        hipDeviceSynchronize();
        dKernelTime += sdkGetTimerValue(&timer);      
    }
    return ((dKernelTime/1000.)/(double)iterations);
}

static void varyEuclidean(){
    static float factor = 1.01f;
    if (euclidean_delta > MAX_EUCLIDEAN_D)
    {
        factor = 1/1.01f;
    }

    if (euclidean_delta < MIN_EUCLIDEAN_D)
    {
        factor = 1.01f;
    }
    euclidean_delta *= factor;
}

static void computeFPS(){
    fpsCount++;
    if (fpsCount == fpsLimit)
    {
        char fps[256];
        float ifps = 1.0f / (sdkGetAverageTimerValue(&timer) / 1000.0f);
        sprintf(fps, "CUDA Color Image Filter:fps(%2.f)", ifps);
        glutSetWindowTitle(fps);
        fpsCount = 0;
        fpsLimit = (int)MAX(ifps, 1.0f);
        sdkResetTimer(&timer);
    }

    if (!g_bInteractive)
    {
        varyEuclidean();
    }
}

// Display
static void display(){
    sdkStartTimer(&timer);
    unsigned int *dResult;
    hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
    hipStreamSynchronize(0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&dResult, &num_bytes, cuda_pbo_resource);
    imageFilterRGBA(dResult, width, height, euclidean_delta, filter_radius, iterations, kernel_timer);
    hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
    hipStreamSynchronize(0);

    {
        //Clear window
        glClearColor (0.0, 0.0, 0.0, 0.0);
        glClearDepth(1.0f);
        glEnable(GL_DEPTH_TEST);
        glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
        glEnable(GL_TEXTURE_2D);
        glEnable(GL_FRAGMENT_PROGRAM_ARB);
        glMatrixMode(GL_MODELVIEW);
        glLoadIdentity();
        
        //  texture from pbo
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST); 
        glBindTexture(GL_TEXTURE_2D, texid);
        glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_MODULATE); 
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
        
        // fragment program 
        glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, shader);
        glEnable(GL_FRAGMENT_PROGRAM_ARB);
        glDisable(GL_DEPTH_TEST);
        
        //Draw textures
        glPushMatrix();
        glBegin(GL_QUADS);
        {
            glTexCoord2f(0, 0); glVertex2f(0, 0);
            glTexCoord2f(1, 0); glVertex2f(1, 0);
            glTexCoord2f(1, 1); glVertex2f(1, 1);
            glTexCoord2f(0, 1); glVertex2f(0, 1);
        }
        glEnd();
        glPopMatrix();
        glBindTexture(GL_TEXTURE_TYPE, 0);
        glDisable(GL_FRAGMENT_PROGRAM_ARB);
        glDisable(GL_TEXTURE_2D);
        glDisable(GL_DEPTH_TEST); 
        
    }
    glutSwapBuffers();
    glutReportErrors();
    sdkStopTimer(&timer);
    computeFPS();
    glutPostRedisplay();
}

//Key board interactive
static void keyboard(unsigned char key, int x, int y){
    if(x!=y) {x=y=0;}    
    switch (key)
    {
        case 27:
                glutDestroyWindow(glutGetWindow());
                return;
            break;

        case 'a':
        case 'A':
            g_bInteractive = !g_bInteractive;
            printf("> Animation is %s\n", !g_bInteractive ? "ON" : "OFF");
            break;

        case ']':
            iterations++;
            break;

        case '[':
            iterations--;
            if (iterations < 1)
            {
                iterations = 1;
            }
            break;
        case '=':
        case '+':
            filter_radius++;
            if (filter_radius > MAX_FILTER_RADIUS)
            {
                filter_radius = MAX_FILTER_RADIUS;
            }
            updateGaussian(gaussian_delta, filter_radius);
            break;

        case '-':
            filter_radius--;
            if (filter_radius < 1)
            {
                filter_radius = 1;
            }
            updateGaussian(gaussian_delta, filter_radius);
            break;

        case 'E':
            euclidean_delta *= 1.5;
            break;

        case 'e':
            euclidean_delta /= 1.5;
            break;

        case 'g':
            if (gaussian_delta > 0.1)
            {
                gaussian_delta /= 2;
            }
            updateGaussian(gaussian_delta, filter_radius);
            break;

        case 'G':
            gaussian_delta *= 2;
            updateGaussian(gaussian_delta, filter_radius);
            break;

        default:
            break;
    }   
    glutPostRedisplay();
}

//Timer 
static void timerEvent(int value)
{
    if(glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    }
}

//Reshape window
static void reshape(int x, int y)
{
    glViewport(0, 0,(GLint)x, (GLint) y);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0f, 1.0f, 0.0f, 1.0f, 0.0f, 1.0f);
    
}
//Clean Device memory
static void cleanup(){
    sdkDeleteTimer(&timer);
    sdkDeleteTimer(&kernel_timer);    
    hipDestroyTextureObject(rgbaTexdImage);
    hipDestroyTextureObject(rgbaTexdTemp);
    hipFree(dImage);
    hipFree(dTemp);
    free(hImage); 
    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &texid);
    glDeleteProgramsARB(1, &shader);
}

static const char *shader_code =
    "!!ARBfp1.0\n"
    "TEX result.color, fragment.texcoord, texture[0], 2D; \n"
    "END";

GLuint compileASMShader(GLenum program_type, const char *code)
{
    GLuint program_id;
    glGenProgramsARB(1, &program_id);
    glBindProgramARB(program_type, program_id);
    glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB, (GLsizei) strlen(code), (GLubyte *) code);    
    
    return program_id;
}

//Initiate GL resource
static void initGLResources()
{
    //PBO
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width*height*sizeof(GLubyte)*4, hImage, GL_STREAM_DRAW_ARB);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    //TID
    glGenTextures(1, &texid);
    glBindTexture(GL_TEXTURE_2D, texid);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);   
    
    cudaGraphicsResourceSetMapFlags(cuda_pbo_resource, cudaGraphicsMapFlagsWriteDiscard);
    hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo,  cudaGraphicsMapFlagsWriteDiscard);
    shader = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shader_code);
}

static void LoadBMPFile(uchar4 **dst, unsigned int *width,  unsigned int *height, const char *name)
{
    BMPHeader hdr;
    BMPInfoHeader infoHdr;
    int x, y;
    FILE *fd;
    printf("Loading %s...\n", name);

    fd = fopen(name,"rb");
    fread(&hdr, sizeof(hdr), 1, fd);

    if (hdr.type != 0x4D42)
    {
        printf("***BMP load error: bad file format***\n");
        exit(EXIT_SUCCESS);
    }
    fread(&infoHdr, sizeof(infoHdr), 1, fd);

    if (infoHdr.bitsPerPixel != 24)
    {
        printf("***BMP load error: invalid color depth***\n");
        exit(EXIT_SUCCESS);
    }
    if (infoHdr.compression)
    {
        printf("***BMP load error: compressed image***\n");
        exit(EXIT_SUCCESS);
    }
    //Dim of image
    *width  = infoHdr.width;
    *height = infoHdr.height;
    *dst    = (uchar4 *)malloc(*width **height * 4);
    printf("BMP width: %u\n", infoHdr.width);
    printf("BMP height: %u\n", infoHdr.height);
    fseek(fd, hdr.offset - sizeof(hdr) - sizeof(infoHdr), SEEK_CUR);
    
#pragma omp parallel 
    for (y = 0; y < infoHdr.height; y++){
#pragma omp parallel  for
        for (x = 0; x < infoHdr.width; x++){
            (*dst)[(y * infoHdr.width + x)].w = 0;
            (*dst)[(y * infoHdr.width + x)].z = fgetc(fd);
            (*dst)[(y * infoHdr.width + x)].y = fgetc(fd);
            (*dst)[(y * infoHdr.width + x)].x = fgetc(fd);
        }
#pragma omp parallel  for
        for (x = 0; x < (4 - (3 * infoHdr.width) % 4) % 4; x++)
        {
            fgetc(fd);
        }
    }
    if (ferror(fd))
    {
        printf("***Unknown BMP load error.***\n");
        free(*dst);
        exit(EXIT_SUCCESS);
    }
    else
    {
        printf("BMP file loaded successfully!\n");
    }
    fclose(fd);
}

int main(int argc, char **argv)
{
    printf("Starting Application..\n");
    setenv ("DISPLAY", ":0", 0);    
    try{ 
        //Device info.
        devID = findCudaDevice(argc, (const char **)argv);
        int runtimeVersion = 0;   
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);   
        hipRuntimeGetVersion(&runtimeVersion);
        fprintf(stderr,"\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        fprintf(stderr,"  CUDA Runtime Version     :\t%d.%d\n", runtimeVersion/1000, (runtimeVersion%100)/10);
        fprintf(stderr,"  CUDA Compute Capability  :\t%d.%d\n", deviceProp.major, deviceProp.minor);

        //Load Image
        char *image_path = NULL;
        const char *image_filename = "src.bmp";
        image_path = sdkFindFilePath(image_filename, 0);
        LoadBMPFile((uchar4 **)&hImage, &width, &height, image_path);
        printf("Loaded '%s', %d x %d pixels\n", image_path, width, height);

        //Initialization of Glut model
        glutInit(&argc, argv);      
        glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE|GLUT_DEPTH);
        glutInitWindowSize(width, height);
        glutInitWindowPosition(100,100);
        glutCreateWindow("CUDA Bilateral Filter");
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutReshapeFunc(reshape);
        glutTimerFunc(REFRESH_DELAY, timerEvent, 0); 
        //Call methods
        updateGaussian(gaussian_delta, filter_radius);
        initTexture(width, height, hImage);
        sdkCreateTimer(&timer);
        sdkCreateTimer(&kernel_timer);
        initGLResources();
        //Instruction
        printf("Running Standard Demonstration with GLUT loop...\n");
        printf("Press '+' and '-' to change filter width\n"
            "Press ']' and '[' to change number of iterations\n"
            "Press 'e' and 'E' to change Euclidean delta\n"
            "Press 'g' and 'G' to change Gaussian delta\n"
            "Press 'a' or  'A' to change Animation mode ON/OFF\n");
        
        glutCloseFunc(cleanup);
        glutMainLoop();        
    }catch(exception &erb){
        std::cerr<<"Error found"<<erb.what()<<"\n";
    }
    return EXIT_SUCCESS;   
}
